#include "hip/hip_runtime.h"
#include <SPH/surface/surfaceDistance.cuh>
#include <utility/include_all.h>

// THIS CODE IS DEFUNCT, DO NOT USE IT.


neighFunctionType estimateSurface(SPH::distance::Memory arrays) {
  //checkedParticleIdx(i);
  //cache_arrays((pos, position), (vol, volume));

  //SWH2<SPH::distance::Memory> swh(arrays, pos[i], vol[i]);
  //auto x_i = pos[i];
  //auto POS = swh.POS;
  //auto normal = POS;
  //auto distance = float_u<SI::m>{ - POS.val.w};
  //auto xW = x_i + distance * normal;
  //auto lutKernel = swh.spline4() / basicVolume;
  //auto lutVal = swh.xBar();

  //auto xBar_t = -normal * lutVal.first + xW * lutVal.second;
  //auto rBar_t = arrays.radius * lutKernel;
  //auto kernelSum = lutKernel;
  //auto numNeighs = swh.count() * 2.f;

  //iterateNeighbors(j) 
  //{
  //  auto w_ij = W_ij;
  //  if (w_ij > 0.f) 
  //  {
  //    kernelSum += w_ij;
  //    xBar_t += pos[j] * w_ij;
  //    rBar_t += math::power<ratio<1, 3>>(vol[j] * PI4O3_1) * w_ij;
  //    numNeighs += 1;
  //  }
  //}

  //auto xBar = xBar_t / kernelSum;
  //auto rBar = rBar_t / kernelSum;

  //auto phi = math::length3(pos[i] - xBar) - rBar;
  //auto r = math::power<ratio<1, 3>>(vol[i] * PI4O3_1);

  //auto phiOld = arrays.distanceBuffer.first[i];

  //phi = math::min(phi, -0.85f * r);

  //if (numNeighs < 15)
  //  phi = -0.85f * r;
  //if (numNeighs > arrays.surface_neighborLimit)
  //  phi = arrays.surface_levelLimit;
  //if (numNeighs > kernelNeighbors() * 1.2f)
  //  phi = arrays.surface_levelLimit;

  //phi = math::clamp(phi, phiOld - 1.0f * r, phiOld + 1.0f * r);

  //iterateBoundaryPlanes(E) {
  //  auto dist = math::planeDistance(E, pos[i]);
  //  if ((dist.val < math::unit_get<1>(arrays.surface_distanceFieldDistances).val &&
  //       fabsf(math::dot3(E, float4_u<>{1.f, 0.f, 0.f, 0.f}).val) > 0.5f) ||
  //      (dist.val < math::unit_get<2>(arrays.surface_distanceFieldDistances).val &&
  //       fabsf(math::dot3(E, float4_u<>{0.f, 1.f, 0.f, 0.f}).val) > 0.5f)) {
  //    phi = arrays.surface_levelLimit;
  //  }
  //}

  //arrays.distanceBuffer.second[i] = phi;
  //arrays.distanceBuffer.first[i] = phiOld;
}

basicFunctionType correctEstimate(SPH::distance::Memory arrays) {
  //checkedParticleIdx(i);
  //auto r = math::power<ratio<1, 3>>(arrays.volume[i] * PI4O3_1);

  //auto phi = arrays.distanceBuffer.second[i];
  //arrays.distanceBuffer.second[i] = phi;
  //arrays.decisionBuffer[i] = phi >= -0.85f * r ? 1.f : 0.f;
  //arrays.markerBuffer[i] = phi >= -0.85f * r ? 1.f : 0.f;

  //if (arrays.markerBuffer[i] < 0.4f) {
  //  arrays.surface_idxBuffer.second[i] = i;
  //}
}
neighFunctionType propagateSurface(SPH::distance::Memory arrays, int32_t threads) {
  //checkedThreadIdx(t);
  //alias_arrays((pos, position));
  //int32_t i = arrays.surface_idxBuffer.first[t];
  //if (i == INT_MIN)
  //  return;

  //int32_t partnerIdx = INT_MAX;
  //float_u<SI::m> partnerDistance{FLT_MAX};
  //auto partnerPhi = 0.0_m;
  ////auto r = math::power<ratio<1, 3>>(arrays.volume[i] * PI4O3_1);
  //arrays.markerBuffer[i] = arrays.decisionBuffer[i];
  //float marker = arrays.markerBuffer[i];

  //iterateNeighbors(j) {
  //  if (W_ij > 0.f) {
  //    if (j == i)
  //      continue;
  //    float neighbor_decision = arrays.decisionBuffer[j];
  //    if (neighbor_decision > 0.2f && marker < 0.05f) {
  //      auto dist = math::abs(math::distance3(pos[i], pos[j]));
  //      if (dist < partnerDistance) {
  //        partnerIdx = j;
  //        partnerDistance = dist;
  //        partnerPhi = arrays.distanceBuffer.second[j];
  //      }
  //    }
  //  }
  //  if (partnerIdx != INT_MAX) {
  //    if (arrays.decisionBuffer[i] < 0.4f) {
  //      auto phi = partnerPhi - partnerDistance;
  //      bool changed = phi > arrays.surface_levelLimit.val * 2.f * arrays.radius;
  //      if (arrays.distanceBuffer.second[i] != phi && changed) {
  //        cuda_atomic<float> change(arrays.changeBuffer);
  //        change.add(1.f);
  //        arrays.distanceBuffer.second[i] = phi;
  //      }
  //      arrays.markerBuffer[i] = changed ? 0.5f : 0.1f;
  //    }
  //  }
  //}
  //if (arrays.markerBuffer[i] < 0.4f) {
  //  arrays.surface_idxBuffer.second[t] = i;
  //} else {
  //  arrays.surface_idxBuffer.second[t] = 0xFFFFFFFF;
  //}
}
neighFunctionType phiSmooth(SPH::distance::Memory arrays) {
  //checkedParticleIdx(i);
  //cache_arrays((pos, position), (vol, volume));

  //arrays.markerBuffer[i] = arrays.decisionBuffer[i];

  //auto phiSum = 0.0_m;
  //auto counter = 0.f;
  //iterateNeighbors(j) {
  //  counter++;
  //  phiSum += arrays.distanceBuffer.second[j] * W_ij * vol[j]; // / arrays.density[neigh];
  //}

  //SWH2<SPH::distance::Memory> swh(arrays, pos[i], vol[i]);
  //auto POS = swh.POS;
  //auto r = math::power<ratio<1, 3>>(arrays.volume[i] * PI4O3_1);
  //auto phiOld = arrays.distanceBuffer.first[i];

  //if (POS.val.w < 1e20f || counter < 5)
  //  phiSum = arrays.distanceBuffer.second[i];

  //phiSum = math::clamp(phiSum, phiOld - 1.0f * r, phiOld + 1.0f * r);

  //arrays.distanceBuffer.first[i] = math::max(phiSum, arrays.surface_levelLimit);
}
neighFunction(estimate, estimateSurface, "Surface: estimate Distance", caches<float4, float>{});
basicFunction(correct, correctEstimate, "Surface: correct Distance");
neighFunction(propagate, propagateSurface, "Surface: Distance iteration");
neighFunction(smooth, phiSmooth, "Surface: smooth Distance", caches<float4, float>{});

struct is_set {
  hostDeviceInline bool operator()(const int x) { return x != -1; }
};

void SPH::distance::distance(Memory mem) {
	if (mem.num_ptcls == 0) return;
  // TODO: FIXME!
  int32_t diff = 0;
  auto compact_idx = [&]() {
    diff = (int32_t) algorithm::copy_if(arrays::surface_idxBuffer::rear_ptr, arrays::surface_idxBuffer::ptr, mem.num_ptcls, is_set());
    cuda::Memset(mem.surface_idxBuffer.second, 0xFF, sizeof(int32_t) * mem.num_ptcls);
  };
  cuda::Memset(mem.surface_idxBuffer.second, 0xFF, sizeof(int32_t) * mem.num_ptcls);
  launch<estimate>(mem.num_ptcls, mem);
   launch<correct>(mem.num_ptcls, mem);
  compact_idx();
  int32_t it = 0;
  do {
    cuda::Memset(mem.changeBuffer, 0x00, sizeof(float));
    launch<propagate>(diff, mem, diff);
    cuda::memcpy(&mem.surface_phiChange, mem.changeBuffer, sizeof(float), hipMemcpyDeviceToHost);
    cuda::memcpy(mem.decisionBuffer, mem.markerBuffer, sizeof(float) * mem.num_ptcls);
    it++;
    if (it % 4 == 0)
      compact_idx();
  } while (mem.surface_phiChange >= 0.5f);
  *parameters::surface_iterations::ptr = it;
  launch<smooth>(mem.num_ptcls, mem);
  *parameters::surface_phiMin::unit_ptr = algorithm::reduce_max(mem.distanceBuffer.first, mem.num_ptcls);
  //if (parameters::adaptive::ratio{} > 1.f)
	 // parameters::surface_neighborLimit{} = 25;
}
