#include "hip/hip_runtime.h"
#define NEW_STYLE
#include <SPH/convection/XSPH.cuh>
#include <utility/include_all.h>

// This function implements the artificial viscosity from XSPH, and additionally adds a stronger
// viscosity if a particles lifetime is negative. This is only the case for particles created with
// splitting which results in an additional layer of blending on top of the normal one. Contrary to
// the normal velocity blending this method basically blends the velocity of split particles with
// all surrounding particles which helps dampen the effects caused by density errors on splitting.
neighFunctionType xsph_viscosity(SPH::XSPH::Memory arrays) {
  checkedParticleIdx(i);
  cache_arrays((pos, position), (vol, volume), (vel, velocity.first), (den, density));

  auto beta_i = getBlendWeight(arrays, i);

  auto sum = vel[i];// - arrays.xsph_viscosity * 0.25f * (beta_i)* vel[i] * pV_b * pW_ib * 0.f;

  iterateNeighbors(j) {
    auto beta_j = getBlendWeight(arrays, j); 
    auto beta_ij = 0.5f * (beta_i + beta_j);
    auto viscosity_constant = arrays.xsph_viscosity * (1.f + beta_ij);

    sum += viscosity_constant * vol[j] / (den[j] + den[i]) * 2.f * (vel[j] - vel[i]) * W_ij; 
  }
  arrays.velocity.second[i] = sum;
#ifdef DEBUG_INVALID_PARITLCES
  auto v_old = vel[i].val;
  auto v_add = sum.val;
  if (v_add.x != v_add.x)
	  printf("%s: Invalid particle %d: " _VECSTR " + " _VECSTR "\n",
		  __FUNCTION__, i,
		  _VEC(v_old), _VEC(v_add));
#endif
}

neighFunction(xsphViscosity, xsph_viscosity, "XSPH Viscosity", caches<float4, float4, float, float>{});

void SPH::XSPH::viscosity(Memory mem) {
  launch<xsphViscosity>(mem.num_ptcls, mem);
  info<arrays::velocity>().swap();
}
