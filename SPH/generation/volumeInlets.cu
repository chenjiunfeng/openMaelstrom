#include "hip/hip_runtime.h"
#define NEW_STYLE
#include <SPH/generation/volumeInlets.cuh>
#include <utility/include_all.h>

// Variadic template base case
hostDeviceInline void emit_particle(uint32_t) {}

// This function sets all parameters of a create particle to a default constructed value, if
// something else is desired it should be done in the particleInlet function.
template <typename T, typename... Ts> hostDeviceInline void emit_particle(uint32_t trgIdx, std::pair<T *, T *> arg, Ts... ref) {
  if (arg.second != nullptr && arg.first != nullptr)
    arg.first[trgIdx] = T{};
  emit_particle(trgIdx, ref...);
}

// This function emits all particles from a single emitter based on the seed positions (
// float4_u<SI::m> *positions ) with the given parameters. This function checks if there is a
// particle in close proximity which for hash based methods requires checking all hash collisions of
// a cell as this method takes places after the position integration which means that the particle
// positions of particles might not be equal to the morton code of the cell they occupy. Similarly
// this interpolation is done at the basic resolution for MLM.
cellFunctionType particleInlet(SPH::streamInlet::Memory arrays, 
	int32_t threads, int32_t ptclsEmitted, int32_t *ptclCounter,
	float4_u<SI::m> *positions, float4_u<SI::velocity> velocity, float_u<SI::volume> particleVolume,  float_u<SI::s> delay, float_u<SI::m> distance, float* distanceBuffer, int32_t* type, int32_t* indicator,
	Ts... tup) {
  checkedThreadIdx(i);

  float4_u<SI::m> position = positions[ptclsEmitted + i];
  bool flag = false;

  float_u<SI::m> a = math::power<ratio<1, 3>>(particleVolume) * 1.f;
  iterateAllCells(position, j) {
	  flag = flag || math::distance3(arrays.position[j], position) < a;
	  if (math::distance3(arrays.position[j], position) < a)
		  arrays.velocity[j] = velocity;
  }

  auto p = position;
  auto r = 0.35f * arrays.radius;
  auto POS = planeBoundary::distance(position, particleVolume, arrays);
  if (POS.val.w <= r) {
	  flag = true;
  }
  for (int32_t b = 0; b < arrays.volumeBoundaryCounter; ++b) {
	  auto VOS = volumeBoundary::distance_fn(position, particleVolume, arrays, b);
	  if (VOS.val.w <= r) {
		  //flag = true;
	  }
  }
  if (flag)
    return;
  cuda_atomic<int32_t> counter(ptclCounter);
  int32_t new_idx = counter.add(1);
  emit_particle(new_idx, tup...);
  arrays.lifetime[new_idx] = -delay;
  position.val.w = support_from_volume(particleVolume).val;
  arrays.position[new_idx] = position;

  arrays.velocity[new_idx] = velocity;
  arrays.volume[new_idx] = particleVolume;
  
  if(distanceBuffer != nullptr) distanceBuffer[new_idx] = distance.val;
  if (type != nullptr) type[new_idx] = 0;
  if (indicator != nullptr) indicator[new_idx] = 0;
}

cellFunction(Inlet, particleInlet, "Particle Inlet")

template <typename... Ts, typename... Vs>
auto callInlet(std::tuple<Ts...>, SPH::streamInlet::Memory mem, int32_t num_ptcls, Vs... args) {
  launch<Inlet>(num_ptcls, mem, num_ptcls, args..., std::make_pair((typename Ts::unit_type *)Ts::ptr, (typename Ts::unit_type *)Ts::rear_ptr)...);
}

void SPH::streamInlet::emit(Memory mem) {
  int32_t num_ptcl_sum = 0;
  for (auto &fluidVolume : get<parameters::inletVolumes>()) {
    //float_u<SI::m> radius = fluidVolume.inlet_radius.value * 1.f / powf(get<parameters::adaptive::resolution>() / 8.f, 1.f / 3.f);
	  float_u<SI::m> radius = fluidVolume.inlet_radius.value;
    float_u<SI::volume> particleVolume = PI4O3 * math::power<3>(radius);

    auto num_ptcls = fluidVolume.particles_emitted.value;
    auto v = fluidVolume.emitter_velocity.value;
    float4_u<SI::velocity> vel{v.x, v.y, v.z, v.w};

	auto factor = sinf(get<parameters::simulationTime>() * (HIP_PI_F * 1.f)) * float4_u<SI::velocity>(0.f,5.f,0.f,0.f);
	//vel += factor;

    float_u<SI::s> dur{fluidVolume.duration.value};
    float_u<SI::s> del{fluidVolume.delay.value};

    if (dur < get<parameters::simulationTime>() && dur > 0.0_s || del > get<parameters::simulationTime>()) {
      num_ptcl_sum += num_ptcls;
      continue;
    }

    int32_t old_count = parameters::num_ptcls{};

    if (old_count + num_ptcls >= parameters::max_numptcls{})
      continue;
	//std::cout << "-----------------------------------------------------" << std::endl;
	//std::cout << *parameters::num_ptcls::ptr << std::endl;
	//std::cout << *parameters::num_ptcls::ptr << std::endl;
	//std::cout << old_count << std::endl;
	//std::cout << "######################################################" << std::endl;
	cuda::memcpy(arrays::inletCounter::ptr, parameters::num_ptcls::ptr, sizeof(int32_t), hipMemcpyHostToDevice);
    callInlet(sorting_list, mem, num_ptcls, num_ptcl_sum, arrays::inletCounter::ptr, (float4_u<SI::m> *)arrays::inletPositions::ptr, vel,
              particleVolume, *parameters::adaptive::delay::ptr, *parameters::surfaceDistance::surface_levelLimit::ptr, arrays::surfaceArrays::distanceBuffer::ptr, arrays::particle_type::ptr, arrays::splitIndicator::ptr );

	//std::cout << *parameters::num_ptcls::ptr << std::endl;
	//std::cout << *parameters::num_ptcls::ptr << std::endl;
	//std::cout << old_count << std::endl;
	cuda::sync("Testing");
	cuda::memcpy(parameters::num_ptcls::ptr, arrays::inletCounter::ptr, sizeof(int32_t), hipMemcpyDeviceToHost);
    num_ptcl_sum += num_ptcls;
  }
}
