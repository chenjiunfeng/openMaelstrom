#include "hip/hip_runtime.h"
#define NO_QT
#include <utility/math.h>
#include <render/quadRender/quadRender.h>

#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <sstream>
#include <texture_types.h>
#include <vector_functions.h>
#include <hip/hip_vector_types.h>

//#define M_PI 3.1415926535897932384626422832795028841971f
#define TWO_PI 6.2831853071795864769252867665590057683943f
#define NUDGE_FACTOR 1e-3f // epsilon
#define samps 1            // samples
#define BVH_STACK_SIZE 32

__device__ __constant__ SceneInformation cScene;
surface<void, cudaSurfaceType2D> surfaceWriteOut;

struct Ray {
  float3 orig; // ray origin 
  float3 dir;  // ray direction
  __device__ Ray(float3 o_, float3 d_) : orig(o_), dir(d_) {}
};

enum Refl_t { DIFF, METAL, SPEC, REFR, COAT }; // material types

struct Sphere {
  float rad;
  float3 pos, emi, col;
  Refl_t refl;        
  __device__ float intersect(const Ray &r) const {
    float3 op = pos - r.orig; //
    float t, epsilon = 0.01f;
    float b = math::dot(op, r.dir);
    float disc = b * b - math::dot(op, op) + rad * rad; // discriminant
    if (disc < 0)
      return 0;
    else
      disc = sqrtf(disc);
    return (t = b - disc) > epsilon ? t : ((t = b + disc) > epsilon ? t : 0);
  }
};
struct Box {
	float3 min, max, emi, col;
	Refl_t refl;
	__device__ float intersect(const Ray &worldRay) const {
		float tmin = ((worldRay.dir.x < 0.f ? max.x : min.x) - worldRay.orig.x) / worldRay.dir.x;
		float tmax = ((worldRay.dir.x < 0.f ? min.x : max.x) - worldRay.orig.x) / worldRay.dir.x;
		float tymin = ((worldRay.dir.y < 0.f ? max.y : min.y) - worldRay.orig.y) / worldRay.dir.y;
		float tymax = ((worldRay.dir.y < 0.f ? min.y : max.y) - worldRay.orig.y) / worldRay.dir.y;

		if ((tmin > tymax) || (tymin > tmax))
			return 0.f;
		if (tymin > tmin)
			tmin = tymin;
		if (tymax < tmax)
			tmax = tymax;

		float tzmin = ((worldRay.dir.z < 0.f ? max.z : min.z) - worldRay.orig.z) / worldRay.dir.z;
		float tzmax = ((worldRay.dir.z < 0.f ? min.z : max.z) - worldRay.orig.z) / worldRay.dir.z;

		if ((tmin > tzmax) || (tzmin > tmax))
			return 0.f;
		if (tzmin > tmin)
			tmin = tzmin;
		if (tzmax < tmax)
			tmax = tzmax;
		return (tmin < 0.f && tmax > 0.f) || (tmin > 0.f && tmax > 0.f) ? (tmin < 0.f ? tmax : tmin) : 0.f;
	}
	__device__ float3 normal(const float3& hitPosition) const {
		if (min.x == -FLT_MAX || min.y == -FLT_MAX || min.z == -FLT_MAX) {
			return float3{ min.x == -FLT_MAX ? 0.f : 1.f, min.y == -FLT_MAX ? 0.f : 1.f, min.z == -FLT_MAX ? 0.f : 1.f };
		}
		constexpr auto epsilon = 1e-6f;
		auto c = (min + max) * 0.5f;
		auto prel = hitPosition - c;
		auto d = math::abs((min - max) * 0.5f);
		auto n = math::castTo<int3>(prel / d * (1.f + epsilon));
		auto nc = char3{ static_cast<char>(n.x), static_cast<char>(n.y), static_cast<char>(n.z) };
		auto normal = math::castTo<float3>(nc);
		return normal;
	}
};

__device__ __constant__ Box boxes[] = {
	{{-25.f, -25.f, 96.f},{25.f,25.f, 132.f},{1.f,1.f,1.f}, {0.f,0.f,0.f}, DIFF}
	,{{190.f, -192.f, -192.f},{192.f,192.f, 192.f},{1.f,1.f,1.f}, {0.f,0.f,0.f}, DIFF}
	//,{ {25, -FLT_MAX, -FLT_MAX},{32, FLT_MAX, FLT_MAX},{0.f,0.f,0.f}, {1.f, 1.f, 1.f}, DIFF}
	//,{ {-FLT_MAX, -25.f, -FLT_MAX},{32, FLT_MAX, FLT_MAX},{0.f,0.f,0.f}, {1.f, 1.f, 1.f}, DIFF}
};
__device__ __constant__ Sphere spheres[] = {
	//{16, {192.0f, 192, 192}, {1.f, 1.f, 1.f}, {0.f, 0.f, 0.f}, DIFF},
	{32, {-96, 0, 16}, {0, 0, 0}, {1.f, 1.f, 1.f}, SPEC},
	{32, {-96, -64, 16}, {0, 0, 0}, {0.5f, 0.f, 0.f}, DIFF},
	{32, {-96, 64, 64}, {0, 0, 0}, {1.0f, 1.f, 1.f}, REFR},
	{10000, {50.0f, 40.8f, -1060}, {0.35f, 0.35f, 0.35f}, {0.075f, 0.075f, 0.075f}, DIFF},
	//{10000, {50.0f, 40.8f, -1060}, {0.55, 0.55, 0.55}, {0.175f, 0.175f, 0.175f}, DIFF},
	//{10000, {50.0f, 40.8f, -1060}, {0.f,0.f,0.f}, {0.f,0.f,0.f}, DIFF},

	{100000, {0.0f, 0, -100000.}, {0, 0, 0}, {0.2f, 0.2f, 0.2f}, DIFF},
	{100000, {0.0f, 0, -100000.1}, {0, 0, 0}, {0.3f, 0.3f, 0.3f}, DIFF} };

__device__ bool RayIntersectsBox(const gpuBVH& bvh, const float3 &originInWorldSpace, const float3 &rayInWorldSpace, int boxIdx) {
  float Tnear, Tfar;
  Tnear = -FLT_MAX;
  Tfar = FLT_MAX;

  float2 limits;

#define CHECK_NEAR_AND_FAR_INTERSECTION(c)                                                                             \
  if (rayInWorldSpace.c == 0.f) {                                                                                    \
    if (originInWorldSpace.c < limits.x)                                                                             \
      return false;                                                                                                    \
    if (originInWorldSpace.c > limits.y)                                                                             \
      return false;                                                                                                    \
  } else {                                                                                                             \
    float T1 = (limits.x - originInWorldSpace.c) / rayInWorldSpace.c;                                              \
    float T2 = (limits.y - originInWorldSpace.c) / rayInWorldSpace.c;                                              \
    if (T1 > T2) {                                                                                                     \
      float tmp = T1;                                                                                                  \
      T1 = T2;                                                                                                         \
      T2 = tmp;                                                                                                        \
    }                                                                                                                  \
    if (T1 > Tnear)                                                                                                    \
      Tnear = T1;                                                                                                      \
    if (T2 < Tfar)                                                                                                     \
      Tfar = T2;                                                                                                       \
    if (Tnear > Tfar)                                                                                                  \
      return false;                                                                                                    \
    if (Tfar < 0.f)                                                                                                    \
      return false;                                                                                                    \
  }
  auto lim = bvh.cudaBVHlimits[boxIdx];
   limits = float2{ lim.bottom.x, lim.top.x };
  CHECK_NEAR_AND_FAR_INTERSECTION(x)
   limits = float2{ lim.bottom.y, lim.top.y };
  CHECK_NEAR_AND_FAR_INTERSECTION(y)
   limits = float2{ lim.bottom.z, lim.top.z };
  CHECK_NEAR_AND_FAR_INTERSECTION(z)
  return true;
}

__device__ bool BVH_IntersectTriangles(gpuBVH& bvh, const float3 &origin, const float3 &ray,
                                       unsigned avoidSelf, int &pBestTriIdx, float3 &pointHitInWorldSpace, float &kAB,
                                       float &kBC, float &kCA, float &hitdist, float3 &boxnormal) {
  pBestTriIdx = -1;
  float bestTriDist;
  bestTriDist = FLT_MAX;
  int32_t stack[BVH_STACK_SIZE];
  int32_t stackIdx = 0;
  stack[stackIdx++] = 0;
  while (stackIdx) {
    int32_t boxIdx = stack[stackIdx - 1];
    stackIdx--;
	uint4 data = bvh.cudaBVHindexesOrTrilists[boxIdx];
    if (!(data.x & 0x80000000)) { // INNER NODE
      if (RayIntersectsBox(bvh, origin, ray, boxIdx)) {
        stack[stackIdx++] = data.y; 
        stack[stackIdx++] = data.z; 
        if (stackIdx > BVH_STACK_SIZE) {
          return false;
        }
      }
    } else { 
      for (uint32_t i = data.w; i < data.w + (data.x & 0x7fffffff); i++) {
         int32_t idx = bvh.cudaTriIdxList[i];
        if (avoidSelf == idx)
          continue;
		float4 normal = bvh.cudaTriangleIntersectionData[idx].normal;
		float d = math::sqlength3(normal);
        float k = math::dot3(normal, ray);
        if (k == 0.0f)
          continue; 
        float s = (normal.w - math::dot3(normal, origin)) / k;
        if (s <= 0.0f)
          continue;
        if (s <= NUDGE_FACTOR)
          continue;
        float3 hit = ray * s;
        hit += origin;

		float4 ee1 = bvh.cudaTriangleIntersectionData[idx].e1d1;
        float kt1 = math::dot3(ee1, hit) - ee1.w;
        if (kt1 < 0.0f)
          continue;
		float4 ee2 = bvh.cudaTriangleIntersectionData[idx].e2d2;
        float kt2 = math::dot3(ee2, hit) - ee2.w;
        if (kt2 < 0.0f)
          continue;
		float4 ee3 = bvh.cudaTriangleIntersectionData[idx].e3d3;
        float kt3 = math::dot3(ee3, hit) - ee3.w;
        if (kt3 < 0.0f)
          continue;
        {
          float hitZ = math::sqdistance(origin, hit);
          if (hitZ < bestTriDist) {
            bestTriDist = hitZ;
            hitdist = sqrtf(bestTriDist);
            pBestTriIdx = idx;
            pointHitInWorldSpace = hit;
            kAB = kt1;
            kBC = kt2;
            kCA = kt3;
          }
        }
      }
    }
  }

  return pBestTriIdx != -1;
}


__device__ float3 path_trace(hiprandState *randstate, float3 originInWorldSpace, float3 rayInWorldSpace, int avoidSelf,
	int32_t numBVHs, gpuBVH* sceneBVH) {
  float3 mask = float3{1.0f, 1.0f, 1.0f};
  float3 accucolor = float3{0.0f, 0.0f, 0.0f};

  for (int32_t bounces = 0; bounces < 5; bounces++) {
    int32_t sphere_id = -1;
    int32_t triangle_id = -1;
    int32_t pBestTriIdx = -1;
    int32_t geomtype = -1;
    const Triangle *pBestTri = NULL;
	float3 pointHitInWorldSpace;

    float d = 1e20f;
    float scene_t = 1e20f;
    float3 f = float3{0, 0, 0};
    float3 emit = float3{0, 0, 0};
    float3 x;  // intersection point
    float3 n;  // normal
    float3 nl; // oriented normal
    float3 dw; // ray direction of next path segment
    Refl_t refltype;

    float3 rayorig = float3{originInWorldSpace.x, originInWorldSpace.y, originInWorldSpace.z};
    float3 raydir = float3{rayInWorldSpace.x, rayInWorldSpace.y, rayInWorldSpace.z};

	float numspheres = sizeof(spheres) / sizeof(Sphere);
	for (int32_t i = int32_t(numspheres); i--;) {
		if ((d = spheres[i].intersect(Ray(rayorig, raydir))) && d < scene_t) {
			scene_t = d;
			sphere_id = i;
			geomtype = 1;
		}
	}
	float kAB = 0.f, kBC = 0.f, kCA = 0.f;
	int32_t bvh_idx = -1;
	for (int32_t i = 0; i < numBVHs; ++i) {
		if (!sceneBVH[i].active)
			continue;
		float ktAB = 0.f, ktBC = 0.f, ktCA = 0.f;
		float hitdistance = 1e20f;
		float3 boxnormal = float3{ 0, 0, 0 };
		float3 point;
		BVH_IntersectTriangles(sceneBVH[i], originInWorldSpace, rayInWorldSpace, avoidSelf, pBestTriIdx, point,
			ktAB, ktBC, ktCA, hitdistance, boxnormal);

		if (hitdistance < scene_t && hitdistance > 0.002f) // EPSILON
		{
			pointHitInWorldSpace = point;
			scene_t = hitdistance;
			triangle_id = pBestTriIdx;
			avoidSelf = pBestTriIdx;
			geomtype = 2;
			bvh_idx = i;
			kAB = ktAB;
			kBC = ktBC;
			kCA = ktCA;
		}
	}
	for (int32_t box_id = 0; box_id < int32_t(sizeof(boxes) / sizeof(Box)); box_id++) {
		Box &box = boxes[box_id];
		float d = box.intersect(Ray(rayorig, raydir));
		if (d && d < scene_t) {
			scene_t = d;
			sphere_id = box_id;
			geomtype = 3;
		}
	}


    if (scene_t > 1e20f)
      return float3{0, 0, 0};

    if (geomtype == 1) {
      Sphere &sphere = spheres[sphere_id];
      x = originInWorldSpace + rayInWorldSpace * scene_t;
      n = math::normalize(float3{x.x - sphere.pos.x, x.y - sphere.pos.y, x.z - sphere.pos.z});
      nl = math::dot(n, rayInWorldSpace) < 0 ? n : n * -1;
      f = float3{sphere.col.x, sphere.col.y, sphere.col.z};
      refltype = sphere.refl;
      emit = float3{sphere.emi.x, sphere.emi.y, sphere.emi.z};
      accucolor += (mask * emit);
    } 
	if (geomtype == 3) {
		Box& box = boxes[sphere_id];
		auto x = originInWorldSpace + rayInWorldSpace * scene_t;
		n = box.normal(x);
		nl = math::dot(n, rayInWorldSpace) < 0 ? n : n * -1;
		f = float3{ box.col.x, box.col.y, box.col.z };
		refltype = box.refl;
		emit = float3{ box.emi.x, box.emi.y, box.emi.z };
		accucolor += (mask * emit);
	}

    if (geomtype == 2) {
      pBestTri = &sceneBVH[bvh_idx].pTriangles[triangle_id];
      x = pointHitInWorldSpace;
      n = math::normalize(math::castTo<float3>(pBestTri->normal));
	  auto i0 = pBestTri->i0;
	  auto i1 = pBestTri->i1;
	  auto i2 = pBestTri->i2;

	  auto v0 = math::castTo<float3>(sceneBVH[bvh_idx].vertices[i0].position);
	  auto v1 = math::castTo<float3>(sceneBVH[bvh_idx].vertices[i1].position);
	  auto v2 = math::castTo<float3>(sceneBVH[bvh_idx].vertices[i2].position);
	  auto n0 = math::castTo<float3>(sceneBVH[bvh_idx].vertices[i0].normal);
	  auto n1 = math::castTo<float3>(sceneBVH[bvh_idx].vertices[i1].normal);
	  auto n2 = math::castTo<float3>(sceneBVH[bvh_idx].vertices[i2].normal);

	  auto ab = v1 - v0;
	  auto bc = v2 - v1;
	  auto cross_ab_bc = math::cross(ab, bc);
	  auto area = math::length(cross_ab_bc);

	  auto ABx = kAB * math::distance(v0, v1);
	  auto BCx = kBC * math::distance(v1, v2);
	  auto CAx = kCA * math::distance(v2, v0);

	  n0 *= BCx / area;
	  n1 *= CAx / area;
	  n2 *= ABx / area;

	  n = math::normalize(n0 + n1 + n2);
	  //return n;

	 // n = math::normalize(math::castTo<float3>(kBC * n0 + kCA * n1 + kAB * n2));

      nl = math::dot(n, rayInWorldSpace) < 0 ? n : n * -1;
      float3 colour = float3{0.9f, 0.3f, 0.0f};
	  if (bvh_idx == 0) {
		  colour = float3{ 0.05098f, 0.23137f, 0.494177f };
		  refltype = REFR;
		  //refltype = COAT;
	  }
	  if (bvh_idx == 1) {
		  colour = float3{ 0.9f, 0.9f, 0.9f };
		  refltype = DIFF;
	  }
      f = colour;
      emit = float3{0, 0, 0};
      accucolor += (mask * emit);
    }

    if (refltype == DIFF) {
      float phi = 2 * HIP_PI_F * hiprand_uniform(randstate);
      float r2 = hiprand_uniform(randstate);
      float r2s = sqrtf(r2);
      float3 w = math::normalize(nl);
      float3 u = math::normalize(math::cross((fabs(w.x) > .1f ? float3{0, 1, 0} : float3{1, 0, 0}), w));
      float3 v = math::cross(w, u);

      dw = math::normalize(u * cosf(phi) * r2s + v * sinf(phi) * r2s + w * sqrtf(1 - r2));
      pointHitInWorldSpace = x + w * 0.01f;
      mask *= f;
    }
    if (refltype == METAL) {
      float phi = 2.f * HIP_PI_F * hiprand_uniform(randstate);
      float r2 = hiprand_uniform(randstate);
      float phongexponent = 20;
      float cosTheta = powf(1 - r2, 1.0f / (phongexponent + 1));
      float sinTheta = sqrtf(1 - cosTheta * cosTheta);
      float3 w = math::normalize(rayInWorldSpace - n * 2.0f * math::dot(n, rayInWorldSpace));
      float3 u = math::normalize(math::cross((fabs(w.x) > .1f ? float3{0, 1, 0} : float3{1, 0, 0}), w));
      float3 v = math::cross(w, u);

      dw = math::normalize(u * cosf(phi) * sinTheta + v * sinf(phi) * sinTheta + w * cosTheta);
      pointHitInWorldSpace = x + w * 0.01f;
      mask *= f;
    }
    if (refltype == SPEC) {
      dw = rayInWorldSpace - n * 2.0f * math::dot(n, rayInWorldSpace);
      pointHitInWorldSpace = x + nl * 0.01f;
      mask *= f;
    }
    if (refltype == COAT) {
      float rouletteRandomFloat = hiprand_uniform(randstate);
      float threshold = 0.05f;
      float3 specularColor = float3{1, 1, 1}; // hard-coded
      bool reflectFromSurface = (rouletteRandomFloat < threshold);

      if (reflectFromSurface) {
        mask *= specularColor;
        dw = rayInWorldSpace - n * 2.0f * math::dot(n, rayInWorldSpace);
        pointHitInWorldSpace = x + nl * 0.01f;
      } else {
        float r1 = 2.f * HIP_PI_F * hiprand_uniform(randstate);
        float r2 = hiprand_uniform(randstate);
        float r2s = sqrtf(r2);
        float3 w = math::normalize(nl);
        float3 u = math::normalize(math::cross((fabs(w.x) > .1f ? float3{0, 1, 0} : float3{1, 0, 0}), w));
        float3 v = math::cross(w, u);

        dw = math::normalize(u * cosf(r1) * r2s + v * sinf(r1) * r2s + w * sqrtf(1 - r2));
        pointHitInWorldSpace = x + nl * 0.01f;
        mask *= f;
      }
    }
    if (refltype == REFR) {
      bool into = math::dot(n, nl) > 0;
      float nc = 1.0f;
      float nt = 1.5f;
      float nnt = into ? nc / nt : nt / nc;
      float ddn = math::dot(rayInWorldSpace, nl);
      float cos2t = 1.0f - nnt * nnt * (1.f - ddn * ddn);
      if (cos2t < 0.0f) {
        dw = rayInWorldSpace;
        dw -= n * 2.0f * math::dot(n, rayInWorldSpace);
        pointHitInWorldSpace = x + nl * 0.01f;
      } else {
        float3 tdir = rayInWorldSpace * nnt;
        tdir -= n * ((into ? 1 : -1) * (ddn * nnt + sqrtf(cos2t)));
        tdir = math::normalize(tdir);

        float R0 = (nt - nc) * (nt - nc) / (nt + nc) * (nt + nc);
        float c = 1.f - (into ? -ddn : math::dot(tdir, n));
        float Re = R0 + (1.f - R0) * c * c * c * c * c;
        float Tr = 1 - Re; // Transmission
        float P = .25f + .5f * Re;
        float RP = Re / P;
        float TP = Tr / (1.f - P);

        if (hiprand_uniform(randstate) < 0.25f) {
          mask *= RP;
          dw = rayInWorldSpace;
          dw -= n * 2.0f * math::dot(n, rayInWorldSpace);

          pointHitInWorldSpace = x + nl * 0.01f;
        } else {
          mask *= TP;
          dw = tdir;
          pointHitInWorldSpace = x + nl * 0.001f;
        }
      }
    }
    originInWorldSpace = pointHitInWorldSpace;
    rayInWorldSpace = dw;
  }
  return float3{accucolor.x, accucolor.y, accucolor.z};
}

__global__ void CoreLoopPathTracingKernel(float3 *accumbuffer, int32_t numBVHs, gpuBVH* sceneBVH, unsigned int framenumber,
                                          unsigned int hashedframenumber) {
  int32_t x = blockIdx.x * blockDim.x + threadIdx.x;
  int32_t y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= cScene.width)
    return;
  if (y >= cScene.height)
    return;

  int32_t threadId =
      (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

  hiprandState randState;
  hiprand_init(hashedframenumber + threadId, 0, 0, &randState);

  float3 rendercampos = float3{cScene.m_camera.position.x, cScene.m_camera.position.y, cScene.m_camera.position.z};

  int32_t i = (cScene.height - y - 1) * cScene.width + x;
  int32_t pixelx = x;
  int32_t pixely = cScene.height - y - 1;

  float3 finalcol = float3{0.0f, 0.0f, 0.0f};
  for (int32_t s = 0; s < 1; s++) {
    float3 rendercamview =
        math::normalize(float3{cScene.m_camera.view.x, cScene.m_camera.view.y, cScene.m_camera.view.z});
    float3 rendercamup = math::normalize(float3{cScene.m_camera.up.x, cScene.m_camera.up.y, cScene.m_camera.up.z});
    float3 horizontalAxis = math::normalize(math::cross(rendercamview, rendercamup));
    float3 verticalAxis = math::normalize(math::cross(horizontalAxis, rendercamview));

    float3 middle = rendercampos + rendercamview;
    float3 horizontal = horizontalAxis * tanf(cScene.m_camera.fov.x * 0.5f * (HIP_PI_F / 180));
    float3 vertical = -verticalAxis * tanf(-cScene.m_camera.fov.y * 0.5f * (HIP_PI_F / 180));

    float jitterValueX = hiprand_uniform(&randState) - 0.5f;
    float jitterValueY = hiprand_uniform(&randState) - 0.5f;
    float sx = (jitterValueX + pixelx) / (cScene.width - 1);
    float sy = (jitterValueY + pixely) / (cScene.height - 1);

    // compute pixel on screen
    float3 pointOnPlaneOneUnitAwayFromEye = middle + (horizontal * ((2 * sx) - 1)) + (vertical * ((2 * sy) - 1));
    float3 pointOnImagePlane =
        rendercampos + ((pointOnPlaneOneUnitAwayFromEye - rendercampos) * cScene.m_camera.focalDistance);

    float3 aperturePoint;
    if (cScene.m_camera.apertureRadius > 0.00001f) {
      float random1 = hiprand_uniform(&randState);
      float random2 = hiprand_uniform(&randState);
      float angle = TWO_PI * random1;
      float distance = cScene.m_camera.apertureRadius * sqrtf(random2);
      float apertureX = cos(angle) * distance;
      float apertureY = sin(angle) * distance;

      aperturePoint = rendercampos + (horizontalAxis * apertureX) + (verticalAxis * apertureY);
    } else {
      aperturePoint = rendercampos;
    }
    float3 apertureToImagePlane = pointOnImagePlane - aperturePoint;
    apertureToImagePlane = math::normalize(apertureToImagePlane);
    float3 rayInWorldSpace = math::normalize(apertureToImagePlane);
    float3 originInWorldSpace = aperturePoint;

    finalcol += path_trace(&randState, originInWorldSpace, rayInWorldSpace, -1, numBVHs, sceneBVH) * (1.0f / samps);
  }
  accumbuffer[i] += finalcol;
  float3 tempcol = accumbuffer[i] / framenumber;
  float3 colour = float3{math::clamp(tempcol.x, 0.0f, 1.0f), math::clamp(tempcol.y, 0.0f, 1.0f),
                         math::clamp(tempcol.z, 0.0f, 1.0f)};
  float4 out{(powf(colour.x, 1 / 2.2f)), (powf(colour.y, 1 / 2.2f)), (powf(colour.z, 1 / 2.2f)), 1.f};
  surf2Dwrite(out, surfaceWriteOut, x * sizeof(float4), y, hipBoundaryModeClamp);
}

void cudaRender(SceneInformation scene, hipGraphicsResource_t resource, objectLoader &sceneMeshes, objectLoader &fluidMeshes, float3 *acc,
                unsigned framenumber, unsigned hashedframes) {
  static bool once = true;
  static gpuBVH* bvhs = nullptr;
  if (once) {
    hipArray_t color_arr;
    hipGraphicsMapResources(1, &resource, 0);
    hipGraphicsSubResourceGetMappedArray(&color_arr, resource, 0, 0);
    cudaBindSurfaceToArray(surfaceWriteOut, color_arr);

	hipMalloc(&bvhs, sizeof(gpuBVH) * 2);
    once = false;
  }
  hipMemcpyToSymbol(HIP_SYMBOL(cScene), &scene, sizeof(SceneInformation));
  dim3 texturedim((uint32_t)scene.width, (uint32_t)scene.height, 1);
  dim3 blockdim(16, 16, 1);
  dim3 griddim(texturedim.x / blockdim.x, texturedim.y / blockdim.y, 1);
  if (texturedim.x % blockdim.x != 0)
    griddim.x += 1;
  if (texturedim.y % blockdim.y != 0)
    griddim.y += 1;

  gpuBVH bvhs_host[] = { fluidMeshes.getGPUArrays(),  sceneMeshes.getGPUArrays() };
  hipMemcpy(bvhs, bvhs_host, sizeof(gpuBVH) * 2, hipMemcpyHostToDevice);

  CoreLoopPathTracingKernel<<<griddim, blockdim>>>((float3 *)acc, 2, bvhs, framenumber,
                                                   hashedframes);
  hipDeviceSynchronize();
}